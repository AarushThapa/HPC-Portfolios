#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <crypt.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


/******************************************************************************

    nvcc -o cuda_password cuda_passwordd.cu -lcrypt


    ./CrackAZ99-With-Data > results.txt


******************************************************************************/


__device__ int match(char *attempt){
	
	char pw1[] = "CV7812";
	char pw2[] = "FT8112";
	char pw3[] = "HS2112";
	char pw4[] = "SC6312";

	char *a = attempt;
	char *b = attempt;
	char *c = attempt;
	char *d = attempt;

	char *p1 = pw1;
	char *p2 = pw2;
	char *p3 = pw3;
	char *p4 = pw4;


while (*a == *p1){
	if(*a == '\0'){
		//printf("Password: %s\n", pw1);
		break;
	}
	a++;
	p1++;
}

while (*b == *p2){
	if(*b == '\0'){
		//printf("Password: %s\n", pw2);
		break;
	}
	b++;
	p2++;
}

while (*c == *p3){
	if(*c == '\0'){
		//printf("Password: %s\n", pw3);
		break;
	}
	c++;
	p3++;
}

while (* d== *p4){
	if(*d == '\0'){
		//printf("Password: %s\n", pw4);
		break;
	}
	d++;
	p4++;
}
return 0;

}

__global__ void kernel(){
	char k1,k2,k3,k4;
	
	char password[7];
	password[6] = '\0';

	int i = blockIdx.x+65;
	int j = threadIdx.x+65;

	char matchone = i;
	char matchtwo = j;
	password[0] = matchone;
	password[1] = matchtwo;
	
	for(k1='0';k1<='9';k1++){
		for(k2='0';k2<='9';k2++){
			for(k3='0';k3<='9';k3++){
				for(k4='0';k4<='9';k4++){
					password[2] = k1;
					password[3] = k2;
					password[4] = k3;
					password[5] = k4;
					if(match(password)){
											
					}else{
						
					}
					
	}
	}
	}
	}

}



int time_difference(struct timespec *start, struct timespec *finish,
                    long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec; 
  long long int dn =  finish->tv_nsec - start->tv_nsec; 

  if(dn < 0 ) {
    ds--;
    dn += 1000000000; 
  } 
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}



int main(int argc, char *argv[]){
  struct timespec start, finish;   
  long long int time_elapsed;

  clock_gettime(CLOCK_MONOTONIC, &start);

 	kernel <<<26,26>>>();
	hipDeviceSynchronize();

  clock_gettime(CLOCK_MONOTONIC, &finish);
  time_difference(&start, &finish, &time_elapsed);
  printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed,
         (time_elapsed/1.0e9)); 

  return 0;
}
