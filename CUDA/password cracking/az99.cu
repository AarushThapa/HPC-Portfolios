#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
/****************************************************************************

  nvcc -o cudaaz99 az99.cu 


     To Run:
     ./cudaaz99 > cudares.txt

 
*****************************************************************************/
__device__ int match(char *attempt) {
  char pw1[] = "CV78";
  char pw2[] = "FT81";
  char pw3[] = "HS21";
  char pw4[] = "SC63";

  char *a = attempt;
  char *b = attempt;
  char *c = attempt;
  char *d = attempt;

  char *p1 = pw1;
  char *p2 = pw2;
  char *p3 = pw3;
  char *p4 = pw4;

  while(*a == *p1) {
   if(*a == '\0')
    {
    printf("Password: %s\n",pw1);
      break;
    }

    a++;
    p1++;
  }
    
  while(*b == *p2) {
   if(*b == '\0')
    {
    printf("Password: %s\n",pw2);
      break;
}

    b++;
    p2++;
  }

  while(*c == *p3) {
   if(*c == '\0')
    {
    printf("Password: %s\n",pw3);
      break;
    }

    c++;
    p3++;
  }

  while(*d == *p4) {
   if(*d == '\0')
    {
    printf("Password: %s\n",pw4);
      break;
    }

    d++;
    p4++;
  }
  return 0;

	}
/****************************************************************************
  The kernel function assume that there will be only one thread and uses
  nested loops to generate all possible passwords and test whether they match
  the hidden password.
*****************************************************************************/

__global__ void  kernel() {
char k1,k2;
 
  char pass[5];
  pass[4] = '\0';

int i = blockIdx.x+65;
int j = threadIdx.x+65;
char matchone = i;
char matchtwo = j;
    
pass[0] = matchone;
pass[1] = matchtwo;

    for(k1='0'; k1<='9'; k1++){
      for(k2='0'; k2<='9'; k2++){
            pass[2] = k1;
            pass[3] = k2;
          if(match(pass)) {
        	//printf("Password Cracked");
          }
             else {
        	 //printf("tried: %s\n", pass);          
       	     }
          }
        } 
      
}
int time_difference(struct timespec *start,
                    struct timespec *finish,
                    long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec;
  long long int dn =  finish->tv_nsec - start->tv_nsec;

  if(dn < 0 ) {
    ds--;
    dn += 1000000000;
  }
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}


int main() {

  struct  timespec start, finish;
  long long int time_elapsed;
  clock_gettime(CLOCK_MONOTONIC, &start);

kernel <<<26,26>>>();
  hipDeviceSynchronize();

  clock_gettime(CLOCK_MONOTONIC, &finish);
  time_difference(&start, &finish, &time_elapsed);
  printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9));
  return 0;
}




